// Final project EEC289Q - Winter 2018
// Applying Recursive Spoke Darts on GPU using CUDA
//https://www.sciencedirect.com/science/article/pii/S1877705816333380

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

#include "kdtree.h"
#include "utilities.h"
#include "RSD_imp.cu"
#include "validate.h"


int main(int argc, char**argv){
	//0) Generate the input points
	PointsGen("../../data/tiny.txt", 100);

	DeviceQuery();
	

	//1) Read input set of points
	int NumPoints;
	real3* Points=NULL;
	ReadPoints("../../data/tiny.txt",NumPoints, Points);


	//2) Build Data Structure
	kdtree tree; 
	uint32_t* h_neighbors;
	int MaxOffset = 32;
	tree.bulkBuild(Points, NumPoints);
	BuildNeighbors(tree, NumPoints, h_neighbors, MaxOffset);
	//TestTree(tree, NumPoints);
	
	//3) Move Data to GPU
	real3* d_points = NULL; uint32_t* d_neighbors = NULL; uint32_t* d_delaunay = NULL;
	HANDLE_ERROR(hipMalloc((void**)&d_delaunay, NumPoints * MaxOffset * sizeof(uint32_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_points, NumPoints * sizeof(real3)));
	HANDLE_ERROR(hipMemcpy(d_points, Points, NumPoints * sizeof(real3), hipMemcpyHostToDevice));	
	HANDLE_ERROR(hipMalloc((void**)&d_neighbors, NumPoints * MaxOffset * sizeof(uint32_t)));
	HANDLE_ERROR(hipMemcpy(d_neighbors, h_neighbors, NumPoints * MaxOffset * sizeof(uint32_t), hipMemcpyHostToDevice));
	
	//4) Launch kernels and record time
	RSD_Imp << <1, 1 >> > (d_points, d_neighbors, NumPoints, d_delaunay, MaxOffset);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());

	//5) Move results to CPU
	uint32_t* h_delaunay = new uint32_t[NumPoints * MaxOffset];
	HANDLE_ERROR(hipMemcpy(h_delaunay, d_delaunay, NumPoints * MaxOffset * sizeof(uint32_t), hipMemcpyDeviceToHost));


	//6) Check correctness of the construction
	validate(NumPoints, Points, h_neighbors, h_delaunay);

	//7) Release memory


	int dummy = 0;
	std::cin >> dummy;


	hipFree(d_points);
	hipFree(d_neighbors);
	hipFree(d_delaunay);

	delete[] Points;
	delete[] h_neighbors;
	delete[] h_delaunay;
	return 0;
}
