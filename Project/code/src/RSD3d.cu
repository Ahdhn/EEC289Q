// Final project EEC289Q - Winter 2018
// Applying Recursive Spoke Darts on GPU using CUDA
//https://www.sciencedirect.com/science/article/pii/S1877705816333380

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

#include "kdtree.h"
#include "utilities.h"
#include "RSD_imp.cu"



int main(int argc, char**argv){
	//0) Generate the input points
	PointsGen("../../data/tiny.txt", 100);


	DeviceQuery();

	//1) Read input set of points
	int NumPoints;
	real3* Points=NULL;
	ReadPoints("../../data/tiny.txt",NumPoints, Points);


	//2) Build Data Structure
	kdtree tree; 
	uint32_t* h_neighbors;
	int MaxOffset = 32;
	tree.bulkBuild(Points, NumPoints);
	BuildNeighbors(tree, NumPoints, h_neighbors, MaxOffset);
	//TestTree(tree, NumPoints);
	
	//3) Move Data to GPU
	/*real3* d_points = NULL; uint32_t* d_neighbors = NULL; uint32_t* d_delaunay = NULL;
	hipMalloc((void**)&d_delaunay, NumPoints * MaxOffset * sizeof(uint32_t));
	HANDLE_ERROR(hipGetLastError());


	hipMalloc((void**)&d_points, NumPoints * sizeof(real3));
	hipMemcpy(d_points, Points, NumPoints * sizeof(real3), hipMemcpyHostToDevice);
	HANDLE_ERROR(hipGetLastError());

	hipMalloc((void**)&d_neighbors, NumPoints * MaxOffset * sizeof(uint32_t));
	hipMemcpy(d_neighbors, h_neighbors, NumPoints * MaxOffset * sizeof(uint32_t), hipMemcpyHostToDevice);
	HANDLE_ERROR(hipGetLastError());

	//4) Launch kernels and record time
	RSD_Imp << <1, 1 >> > (d_points, d_neighbors, NumPoints, d_delaunay, MaxOffset);
	HANDLE_ERROR(hipGetLastError());
	hipDeviceSynchronize();

	//5) Move results to CPU
	uint32_t* h_delaunay = new uint32_t[NumPoints * MaxOffset];
	hipMemcpy(h_delaunay, d_delaunay, NumPoints * MaxOffset * sizeof(uint32_t), hipMemcpyDeviceToHost);


	//6) Check correctness of the construction


	//7) Release memory


	int dummy = 0;
	std::cin >> dummy;


	hipFree(d_points);
	hipFree(d_neighbors);
	hipFree(d_delaunay);

	delete[] Points;
	delete[] h_neighbors;
	delete[] h_delaunay;*/
	return 0;
}
