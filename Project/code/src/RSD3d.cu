#include "hip/hip_runtime.h"
// Final project EEC289Q - Winter 2018
// Applying Recursive Spoke Darts on GPU using CUDA
//https://www.sciencedirect.com/science/article/pii/S1877705816333380

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <hiprand.h>


#include "kdtree.h"
#include "utilities.h"
#include "RSD_imp.cu"
#include "validate.h"
#include "extractTets.h"

__global__ void initialise_curand_on_kernels(hiprandState * state, unsigned long seed)
{
	//stolen from https://nidclip.wordpress.com/2014/04/02/cuda-random-number-generation/

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

int main(int argc, char**argv){
	//0) Generate the input points
	PointsGen("../../data/tiny.txt", 100);

	DeviceQuery();
	

	//1) Read input set of points
	int NumPoints;
	real3* Points=NULL;
	ReadPoints("../../data/tiny.txt",NumPoints, Points);


	//2) Build Data Structure
	kdtree tree; 
	uint32_t* h_neighbors;
	int MaxOffset = 32;
	tree.bulkBuild(Points, NumPoints);
	BuildNeighbors(tree, NumPoints, h_neighbors, MaxOffset);
	//TestTree(tree, NumPoints);
	
	//3) Move Data to GPU
	real3* d_points = NULL; uint32_t* d_neighbors = NULL; uint32_t* d_delaunay = NULL;
	HANDLE_ERROR(hipMalloc((void**)&d_delaunay, NumPoints * MaxOffset * sizeof(uint32_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_points, NumPoints * sizeof(real3)));
	HANDLE_ERROR(hipMemcpy(d_points, Points, NumPoints * sizeof(real3), hipMemcpyHostToDevice));	
	HANDLE_ERROR(hipMalloc((void**)&d_neighbors, NumPoints * MaxOffset * sizeof(uint32_t)));
	HANDLE_ERROR(hipMemcpy(d_neighbors, h_neighbors, NumPoints * MaxOffset * sizeof(uint32_t), hipMemcpyHostToDevice));
	
	//3.5) initialize rand number generator 
	//srand(time(NULL));
	hiprandState* deviceStates = NULL;
	/*int num = 1;
	HANDLE_ERROR(hipMalloc(&deviceStates, num * sizeof(hiprandState)));
	initialise_curand_on_kernels << <num / 1024 + 1, 1024 >> >(deviceStates, unsigned(time(NULL)));
	HANDLE_ERROR(hipDeviceSynchronize());*/


	//4) Launch kernels and record time
	RSD_Imp << <1, 1 >> > (d_points, d_neighbors, NumPoints, d_delaunay, MaxOffset, deviceStates);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());

	//5) Move results to CPU
	uint32_t* h_delaunay = new uint32_t[NumPoints * MaxOffset];
	HANDLE_ERROR(hipMemcpy(h_delaunay, d_delaunay, NumPoints * MaxOffset * sizeof(uint32_t), hipMemcpyDeviceToHost));


	//6) Check correctness of the construction
	std::vector<std::vector<uint32_t>> myTets = extractTets(NumPoints, h_delaunay, MaxOffset);
	validate(myTets, Points, h_neighbors,MaxOffset);	

	//7) Release memory


	int dummy = 0;
	std::cin >> dummy;


	hipFree(d_points);
	hipFree(d_neighbors);
	hipFree(d_delaunay);
	hipFree(deviceStates);

	delete[] Points;
	delete[] h_neighbors;
	delete[] h_delaunay;
	return 0;
}
