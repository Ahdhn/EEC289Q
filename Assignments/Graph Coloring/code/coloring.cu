#include "hip/hip_runtime.h"
#include "assign_color.cu"
#include "indept_set.cu"
//#include "filter.cu"

#define NUM_COLOR_PER_THREAD 1 //this is changed to be more than one, then we need to move my_offset_start and my_offset_end to be in shared memory instead 

 __device__ int numColored = 0;  

__global__ void coloring(uint32_t NumRow, //number of vertices (= number of rows in adjacency matrix)
	                     uint32_t *col_id, //the column id in the CSR format 
	                     uint32_t *offset, //the row offset in the CSR
	                     unsigned char* color, //the color of the vertices (output)	                     
	                     int*numColors,//the total colors that has been assigned 	                     
	                     uint32_t*numberVerticesPerColor,
	                     uint32_t max_NNZ_per_block
	                     ){
	
	                     

	//Every block will own number of vertices equal to blockDim.x 
	//We need to move the data (col_id, offset) for the block's vertices 
	//The color of each vertex will be stored in a register (since one vertex is handeled by one thread)
	//and will write just once at the end (coalesced write)
	if(threadIdx.x == 0 && blockIdx.x == 0){
		(*numColors)=0;
	}
		
	unsigned char my_thd_colors [NUM_COLOR_PER_THREAD];//the colos that this thread will do
#pragma unroll 	
	for(uint32_t i=0; i < NUM_COLOR_PER_THREAD; i++){ my_thd_colors[i]=0; }//initialize with no color 

	extern __shared__ bool shrd_ptr[];
	bool* sh_set = shrd_ptr;// the independent set of this block (has to be in shared memory because thread/vertex i needs to know if thread/vertex j is in the set or not when getting the independant set)
	//uint32_t * sh_col_id = (uint32_t*)&sh_set[blockDim.x]; 

	/*extern __shared__ uint32_t shrd_ptr[];
	uint32_t * sh_col_id = shrd_ptr;//(uint32_t*)&sh_set[blockDim.x]; 
	bool* sh_set = (bool*)&sh_col_id[max_NNZ_per_block];*/
	

	//extern __shared__ uint32_t shrd_ptr2[];
	//uint32_t * sh_col_id = (uint32_t*)&shrd_ptr[NUM_COLOR_PER_THREAD*blockDim.x]; //has length of number of nnz elements for this block's vertices 	
	//uint32_t * sh_col_id = shrd_ptr2; 
	

	uint32_t my_offset_start, my_offset_end;//the offset of of the row this thread is responsible of 
	                                        //we read the start and end because we need them to iterate over 
	                                        //the columns of this row

	const uint32_t start_row = blockIdx.x * blockDim.x; //starting point of my block (in the offset array) 
	                                                    //starting row in the adjacency matris
	                                                    //this block is resposible of vertices with id from 
	                                                    //start up to start + blockDim.x
	const uint32_t end_row = blockIdx.x * blockDim.x + blockDim.x;

	const uint32_t tid = start_row + threadIdx.x;//equal to this thread's vertex id

	if(tid < NumRow){
		my_offset_start = offset[ tid ];
		my_offset_end = offset[tid +1];	
	}

#pragma unroll 
	for(uint32_t i= threadIdx.x; i< NUM_COLOR_PER_THREAD*blockDim.x; i+=blockDim.x){
		sh_set[i] = false;
	}


	//count the number of nnz element owned by this block
	//reduce within a block 
	__shared__ uint32_t block_myNNZ; 
	__shared__ uint32_t block_start_col_id;


	if(threadIdx.x == blockDim.x -1){
		block_myNNZ = my_offset_end;
	}
	__syncthreads(); //make sure block_myNNZ is updated 
	if(threadIdx.x == 0){
		block_myNNZ -= my_offset_start;
		block_start_col_id = my_offset_start;
	}
	__syncthreads(); //make sure block_myNNZ is updated 
	my_offset_start -= block_start_col_id; //decremented so that we can use them to index sh_col_id directly
	my_offset_end -= block_start_col_id;

	/*if(threadIdx.x == 0){
		printf("\n block_myNNZ= %d\n",block_myNNZ);
	}*/


	//move the col_id (coalesced read)	

	/*for(uint32_t i = threadIdx.x; i < block_myNNZ; i+=blockDim.x){
		
		sh_col_id[i] = col_id[block_start_col_id + i];

		printf("\n tid= %d, blockIdx.x= %d, block_myNNZ= %d, block_start_col_id= %d, threadIdx.x= %d, i= %d, col_id[%d]= %d\n", tid, blockIdx.x, block_myNNZ, block_start_col_id, threadIdx.x, i,block_start_col_id + i,col_id[block_start_col_id + i]);		
		
	}*/
	__syncthreads(); 
	//return;

	//*****************************************************//

	unsigned char currentColor = 1;	

	while(numColored < blockDim.x * NUM_COLOR_PER_THREAD){ //loop untill all this blocks vertices are colored		
		numColored = 0;
		indept_set(tid, my_offset_start, my_offset_end, start_row, end_row, col_id, NumRow, numColored, 0, sh_set, block_start_col_id);
		__syncthreads();			
		assign_color(tid, currentColor, NumRow, sh_set, my_thd_colors, 0);		
		__syncthreads();
		currentColor++;			
	}

	//move color to global memory
	if(tid < NumRow){
		color[tid] = my_thd_colors[0];				
		atomicMax(numColors,int(color[tid]));
		atomicAdd(&numberVerticesPerColor[my_thd_colors[0]], uint32_t(1));
	}
}


/*__device__ int numColored = 0;

__global__ void coloring(uint32_t NumRow, //number of vertices (= number of rows in adjacency matrix)
	                     uint32_t numNNZ, //number of non zero entry of the adjacency matrix
	                     uint32_t *col_id, //the column id in the CSR format 
	                     uint32_t *offset, //the row offset in the CSR
	                     unsigned char* color, //the color of the vertices (output)
	                     bool*set //the indepent set (global memory)
	                     ){

	unsigned char currentColor = 1;
	

	while(numColored < NumRow){//loop untill all vertices are colored 

		indept_set(NumRow, numNNZ, col_id, offset, set, currentColor%2 == 1, color, numColored);
		__syncthreads();		
		assign_color(currentColor, NumRow, set,color);
		__syncthreads();
		//filter();
		currentColor++;	
	}		
}*/