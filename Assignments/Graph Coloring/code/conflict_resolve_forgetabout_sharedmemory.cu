#include "hip/hip_runtime.h"
//#include <cub/block/block_load.cuh>
//#include <cub/block/block_store.cuh>
//#include <cub/block/block_scan.cuh>
//
//using namespace hipcub;
//
#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>

using namespace hipcub;

__global__
void GetneighLen(uint32_t *nodes, int sizeNode,  uint32_t *tr_offset, uint32_t *neighLen, uint32_t *changeColor){
	
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeNode; i+=gridDim.x*blockDim.x)
	{
		neighLen[i] = tr_offset[nodes[i]+1]-tr_offset[nodes[i]];
		changeColor[i] = 0;
	}
}



__global__
void FindChangeColor(uint32_t *changeColor, uint32_t sizeNode, uint32_t *nodes, uint32_t *neighLen, uint32_t sizeNeighLen, uint32_t *tr_col_id, uint32_t *tr_offset) {

        int neighbor=-1;
	int neighborOwner = -1;
        for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeNeighLen; i+=blockDim.x*gridDim.x)
        {
                // calculate who to fetch
                for(int j=0; j<sizeNode; j++)
                {
                        if(i>=neighLen[j] && i<neighLen[j+1])
                        {
                                //j's neighborlist
                                neighborOwner = j;
				break;
                        }
                }

		neighbor = tr_col_id[tr_offset[nodes[neighborOwner]]+(i-neighLen[neighborOwner])];
                                printf("I am thread%d, my neighberOwner=%d nodes[%d]=%d neighbor=%d\n", threadIdx.x, neighborOwner,neighborOwner,nodes[neighborOwner], neighbor);
		for(int j=0; j<sizeNode; j++)
		{
			if(neighbor == nodes[j])
			{
				changeColor[neighborOwner]=1;
				break;
			}
		}
		if(threadIdx.x==0)
		{
			for(int i=0; i<sizeNode; i++)
				printf("changeColor[%d]=%d   ", i,changeColor[i] );
			printf("\n");
		}
        }
}

__global__
void Conflict_assignColor(uint32_t *changeColor, int theColor, unsigned char *color, uint32_t *nodes, uint32_t sizeNode) {
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeNode; i+=blockDim.x*gridDim.x)
	{
		if(changeColor[i]==0)
			color[nodes[i]] = (unsigned char)theColor;
	}
}


__global__
void GenNewNodes(uint32_t *nodes, uint32_t *newNodes, uint32_t *neighLen, uint32_t *newNeighLen, int sizeNode, uint32_t *changeColor) {

	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeNode; i+=blockDim.x*gridDim.x)
	{
		if(changeColor[i]+1 == changeColor[i+1])
		{
			newNodes[changeColor[i]] = nodes[i];
			newNeighLen[changeColor[i]] = neighLen[i];
		}
	}
}




int conflict_resolve_forgetabout_sharedmemory(uint32_t* conflict_color, // Array of conflict vetices grouped by color
                      uint32_t *conflict_color_offset, // offset of different color on conflit_color 
                      uint32_t *tr_col_id, // CSR of graph, but only lower triangle part
                      uint32_t *tr_offset, // CSR offset of graph, but only lower triangle part
                      uint32_t numVertices, // number of vertices
                      uint32_t size_tr_col,// size of tr_col_id
                      uint32_t numColor,// number of color has been used
                      unsigned char *color,// color array for all vertices
		      uint32_t colorID ) // working space and the size of this array shoudl be BLOCK_THREADS*ITEM_PER_THREAD, o,w it overwrittern information
{
	uint32_t *nodes(NULL), *changeColor(NULL), *nodes1(NULL), *nodes2(NULL), *neighLen1(NULL), *neighLen2(NULL), *scanArray(NULL), *neighLen(NULL);
	uint32_t *newNodes(NULL), *newNeighLen(NULL);
	int start = conflict_color_offset[colorID];
	int end = conflict_color_offset[colorID+1];
	int sizeNode = end-start;
	if(sizeNode==0) return numColor;
	std::cout<<"start: "<<start<<" end: "<<end<<" sizeNode: "<<sizeNode <<std::endl;
	nodes = conflict_color+start;
	HANDLE_ERROR(hipMallocManaged(&changeColor, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&nodes1, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&nodes2, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&neighLen1, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&neighLen2, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&scanArray, (sizeNode+1)*sizeof(uint32_t)));
	
	std::cout<<"allocate succeed"<<std::endl;
	for(int i=0; i<sizeNode; i++)
	{
		std::cout<<"nodes["<<i<<"]= "<<nodes[i]<<"  ";
	}
	std::cout<<std::endl;

	GetneighLen<<<1, 32>>>(nodes, sizeNode,  tr_offset, neighLen1, changeColor);
	hipDeviceSynchronize();



	for(int i=0; i<sizeNode; i++)
	{
		std::cout<<"changeColor["<<i<<"]= "<<changeColor[i]<<"  ";
	}
	std::cout<<std::endl;
		for(int i=0; i<numVertices; i++)
		{
			printf("color[%d]=%u  ", i, color[i]);
		}
		std::cout<<std::endl;

	int choseL = 0;
	int counter = 0;
	neighLen = neighLen1;
	int theColor = colorID;
	void  *d_temp_storage = NULL;
    	size_t    temp_storage_bytes = 0;
	DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, neighLen, scanArray, sizeNode+1);
	HANDLE_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));

//	//test
//	uint32_t *test(NULL), *test_out(NULL);
//	HANDLE_ERROR(hipMallocManaged(&test, 10*sizeof(uint32_t)));
//	HANDLE_ERROR(hipMallocManaged(&test_out, 11*sizeof(uint32_t)));
//	for(int i=0; i<10; i++)
//		test[i]=i;
//
//	DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, neighLen, scanArray, 3);
//	HANDLE_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));
//	DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, neighLen, scanArray, 3);
//	hipDeviceSynchronize();
//	for(int i=0; i<2; i++)
//		std::cout<<" "<<neighLen[i]<<" ";
//	std::cout<<std::endl;
//
//	for(int i=0; i<3; i++)
//		std::cout<<" "<<scanArray[i]<<" ";
//	
//	std::cout<<std::endl;
	
	while(true) {
		for(int i=0; i<sizeNode; i++)
		{
			std::cout<<"neighLen["<<i<<"]= "<<neighLen[i]<<"  ";
		}
		std::cout<<std::endl;

		DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, neighLen, scanArray, sizeNode+1);			
		hipDeviceSynchronize();
		for(int i=0; i<sizeNode+1; i++)
		{
			std::cout<<"scan_neighLen["<<i<<"]= "<<scanArray[i]<<"   ";
		}
		std::cout<<std::endl;
		
		FindChangeColor<<<1,32>>>(changeColor, sizeNode, nodes, scanArray, scanArray[sizeNode], tr_col_id, tr_offset);
		hipDeviceSynchronize();

		theColor = numColor+counter;
		if(counter!=0)
			Conflict_assignColor<<<1,32>>>(changeColor, theColor, color, nodes, sizeNode);
		hipDeviceSynchronize();

		for(int i=0; i<numVertices; i++)
		{
			 printf("color[%d]=%u  ", i, color[i]);
		}
		std::cout<<std::endl;

		DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, changeColor, scanArray, sizeNode+1);	
		hipDeviceSynchronize();

		for(int i=0; i<sizeNode; i++)
		{
			std::cout<<"changeColor["<<i<<"]= "<<changeColor[i]<<"  ";
		}
		std::cout<<std::endl;	
		
		for(int i=0; i<sizeNode+1; i++)
		{
			std::cout<<"scan_changeColor["<<i<<"]= "<<scanArray[i]<<"   ";
		}
		std::cout<<std::endl;

		choseL = choseL^1;
		if(choseL == 1)
		{
			newNeighLen = neighLen2;
			newNodes = nodes2;
		}
		else
		{
			newNeighLen = neighLen1;
			newNodes = nodes1;
		}
	
		std::cout<<"sizeNode: "<< sizeNode<<std::endl;
		GenNewNodes<<<1,32>>>(nodes, newNodes, neighLen, newNeighLen, sizeNode, scanArray);
		hipDeviceSynchronize();
		sizeNode = scanArray[sizeNode];
		if(sizeNode == 0) break;

		printf("new sizeNode: %d\n", sizeNode);
		nodes = newNodes;
		for(int i=0; i<sizeNode; i++)
		{
			std::cout<<"newNode["<<i<<"]=  "<<nodes[i]<<"   ";
		}
		std::cout<<std::endl;
		neighLen = newNeighLen;
		for(int i=0; i<sizeNode; i++)
		{
			std::cout<<"newNeighLen["<<i<<"]=  "<<neighLen[i]<<"   ";
		}
		std::cout<<"counter: "<<counter<<std::endl;
		counter++;
	}	
	std::cout<<counter<<" color is added, total number of color is "<<theColor<<std::endl;
	return theColor;
}
