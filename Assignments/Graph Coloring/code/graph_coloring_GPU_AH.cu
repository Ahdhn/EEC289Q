#include "hip/hip_runtime.h"
#include <sstream>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipsolver.h>

//Error handling micro, wrap it around function whenever possible
static void HandleError(hipError_t err, const char *file, int line) {
   if (err != hipSuccess) {
      printf("\n%s in %s at line %d\n", hipGetErrorString(err), file, line);
      //system("pause");
      exit(EXIT_FAILURE);
   }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#include "validate.h"
#include "serial.h"
#include "utility.h"
#include "coloring.cu"
#include "cuda_query.cu"

uint32_t maxNNZ_per_segment(uint32_t*offset, uint32_t NumRow, uint32_t segment_length){
   //count the max number of nonzero elements within a segment_length of offset array
   //i.e., how many nonzero elements between row i and row j such that j-i = segment_length
   uint32_t max_seg = 0;
   

   for(uint32_t seg_start=0; seg_start < NumRow; seg_start+= segment_length) {      
      uint32_t seg_end = (seg_start + segment_length < NumRow ) ? seg_start+segment_length : NumRow;
      uint32_t my_len = offset[seg_end] - offset[seg_start];
      //std::cout<<" seg_start= "<<seg_start<< " seg_end= "<<seg_end<<" my_len= "<<my_len<< " offset[seg_end]= "<< offset[seg_end]<< " offset[seg_start]= "<< offset[seg_start]<<std::endl;
      if (my_len > max_seg){max_seg = my_len;}
   }
   return max_seg;
}

int main(int argc, char* argv[]){
   cuda_query(3); //Set the deivde number here 

   if(argc != 2){
      std::cout<<"  Usage ./graphGPU INPUTFILE"<<std::endl;
      std::cout<<"input files can be found under input/ "<<std::endl;
      exit(EXIT_FAILURE);
   }

   bool* graph;
   int V;  
   uint32_t numNNZ=0;
   uint32_t NumRow=0; //same as V


   //1) Read graph
   if (std::string(argv[1]).find(".col") != std::string::npos){
     ReadColFile(argv[1], &graph, &V, &numNNZ,&NumRow);
   } else if (std::string(argv[1]).find(".mm") != std::string::npos){
     ReadMMFile(argv[1], &graph, &V, &numNNZ,&NumRow);
   } else{
      std::cout<<" Invalid file formate!!"<<std::endl;
      exit(EXIT_FAILURE);
   }
   /***********************************************************************/

   //2) Allocate memory (on both sides)
   uint32_t *col_id(NULL),*offset(NULL);   
   HANDLE_ERROR(hipMallocManaged(&col_id, numNNZ*sizeof(uint32_t)));
   
   //last entry will be = numNonZero (so that we have always a pointer
   //to the first and last id for each row with no need for if statments)   
   HANDLE_ERROR(hipMallocManaged(&offset, (NumRow +1)*sizeof(uint32_t)));
   /***********************************************************************/


   int numBlocks(1), numThreads(1);
   if(NumRow < 1024){//if it is less than 1024 vertex, then launch one block 
      numBlocks = 1;
      numThreads = NumRow;
   }else{//otherwise, launch as many as 1024-blocks as you need      
      numBlocks = (NumRow/1024) + 1;
      numThreads = 1024;
   }

   const uint32_t blockingSize = 2;

   //3) Get graph in CSR format 
   //getCSR(numNNZ, NumRow, graph, col_id, offset);
   getBlockedCSR(numNNZ, NumRow, graph, col_id, offset, blockingSize);
   //printCSR(numNNZ,NumRow,col_id, offset);   
   /***********************************************************************/


   //5) Color Vertices in paralllel
   unsigned char* color;
   HANDLE_ERROR(hipMallocManaged(&color, NumRow*sizeof(unsigned char)));
   memset(color, 0, NumRow );
   bool*set;
   HANDLE_ERROR(hipMallocManaged(&set, NumRow*sizeof(bool)));
   memset(set, 0, NumRow); 

   uint32_t max_NNZ_per_block= maxNNZ_per_segment(offset, NumRow, blockingSize);
   //std::cout<< "  max_NNZ_per_block= "<<max_NNZ_per_block <<std::endl;

   
   const uint32_t shrd_mem = numThreads*sizeof(bool) + max_NNZ_per_block*numThreads*sizeof(uint32_t);
                              //^^ sh_set                     ^^ sh_col_id
   std::cout<<" shrd_mem="<<shrd_mem<<" = numThreads*sizeof(bool){"<<numThreads*sizeof(bool)<<"} + max_NNZ_per_block*numThreads*sizeof(uint32_t){"<<max_NNZ_per_block*numThreads*sizeof(uint32_t)<<"}"<<std::endl;
   coloring <<<numBlocks, numThreads, shrd_mem>>> (NumRow, col_id, offset, color);     
   hipDeviceSynchronize(); 
   /***********************************************************************/

    //5.5) Validate parallel LOCAL solution 
   printf("Parallel LOCAL solution has %d colors\n", CountColors(V, color));
   printf("Valid LOCAL coloring: %d\n\n", IsValidColoring_Blocked(graph, V, color, blockingSize));
   PrintSolution(color,V);
   /***********************************************************************/


   //6) Validate parallel solution 
   //printf("Parallel solution has %d colors\n", CountColors(V, color));
   //printf("Valid coloring: %d\n\n", IsValidColoring(graph, V, color));
   //PrintSolution(color,V);
   /***********************************************************************/


   //7) Color Vertices on CPU
   //GraphColoring(graph, V, &color);
   //printf("Brute-foce solution has %d colors\n", CountColors(V, color));   
   //printf("Valid coloring: %d\n", IsValidColoring(graph, V, color));

   GreedyColoring(graph, V, &color);
   printf("\n*********************************\n");
   printf("Greedy solution has %d colors\n", CountColors(V, color));
   printf("Valid coloring: %d\n\n", IsValidColoring(graph, V, color));
   //PrintSolution(color,V);
   /***********************************************************************/


   //8)Compare solution 
   /***********************************************************************/

   return 0;
}