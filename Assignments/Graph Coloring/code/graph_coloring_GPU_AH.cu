#include "hip/hip_runtime.h"
#include <sstream>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipsolver.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>

//Error handling micro, wrap it around function whenever possible
static void HandleError(hipError_t err, const char *file, int line) {
   if (err != hipSuccess) {
      printf("\n%s in %s at line %d\n", hipGetErrorString(err), file, line);
      //system("pause");
      exit(EXIT_FAILURE);
   }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#include "validate.h"
#include "serial.h"
#include "utility.h"
#include "coloring.cu"
#include "cuda_query.cu"
#include "conflict_resolve.cu"
//#include "conflict_resolve_forgetabout_sharedmemory.cu"
#include "conflict_resolve_forgetabout_sharedmemory1.cu"
#include "conflict_graph.cu"

void GraphColoringGPU(const char filename[], int**&color_int)
{
   bool* graph;
   int V;     
   uint32_t blockingSize = 5;//TODO
   uint32_t numNNZ=0;
   uint32_t NumRow=0; 
   uint32_t numNNZ_blocked = 0;


   //1) Read graph
   if (std::string(filename).find(".col") != std::string::npos){
     ReadColFile(filename, &graph, &V, &numNNZ,&NumRow, blockingSize, &numNNZ_blocked);
   } else if (std::string(filename).find(".mm") != std::string::npos){
     ReadMMFile(filename, &graph, &V, &numNNZ,&NumRow,blockingSize, &numNNZ_blocked);
   } else{
      std::cout<<" Invalid file formate!!"<<std::endl;
      exit(EXIT_FAILURE);
   }      
   /***********************************************************************/  

   //2) Allocate memory (on both sides)
   uint32_t *col_id(NULL),*offset(NULL);   

   HANDLE_ERROR(hipMallocManaged(&col_id, numNNZ_blocked*sizeof(uint32_t)));   
   HANDLE_ERROR(hipMallocManaged(&offset, (NumRow +1)*sizeof(uint32_t)));   


   unsigned char* color;
   HANDLE_ERROR(hipMallocManaged(&color, NumRow*sizeof(unsigned char)));   
   memset(color, 0, NumRow);   
   int*numColor;
   HANDLE_ERROR(hipMallocManaged(&numColor, sizeof(int)));
   memset(numColor, 0, 1);   
   uint32_t*numberVerticesPerColor;//allocate as if each vertex will take different color 
   HANDLE_ERROR(hipMallocManaged(&numberVerticesPerColor, NumRow*sizeof(uint32_t)));
   memset(numberVerticesPerColor, 0, NumRow);   
   /***********************************************************************/
   

   //3) Get graph in CSR format 
   //getCSR(numNNZ, NumRow, graph, col_id, offset);   
   uint32_t maxLeftout=0; //maxLeftout the maximum number of vertices j connected to i that are left out when constructing blocked CSR (used to allocate conflicting graph)
   getBlockedCSR(NumRow, graph, col_id, offset, blockingSize, maxLeftout);   
   
   

   //CUB parameters 
   void *d_temp_storage = NULL;
   size_t temp_storage_bytes = 0;
   hipcub::CachingDeviceAllocator  g_allocator(true);

   int numBlocks(1), numThreads(1);
   numBlocks = (NumRow/blockingSize) + 1;
   numThreads = blockingSize;

   //A) Do local colring 
   uint32_t max_NNZ_per_block= maxNNZ_per_segment(offset, NumRow, blockingSize);        
   uint32_t shrd_mem = numThreads*sizeof(bool) /*+ max_NNZ_per_block*sizeof(uint32_t)*/;  
   std::cout<<"NumRow= "<<NumRow <<"   max_NNZ_per_block= "<<max_NNZ_per_block << " shrd_mem= "<<shrd_mem  << std::endl;
   std::cout<<" numThreads= "<<numThreads<< " numBlocks= "<< numBlocks<<std::endl;
   coloring <<<numBlocks, numThreads, shrd_mem>>> (NumRow, col_id, offset, color, numColor,numberVerticesPerColor, max_NNZ_per_block);
   hipDeviceSynchronize();     

   HANDLE_ERROR(hipFree(offset));//free what you dont need 
   HANDLE_ERROR(hipFree(col_id));

   //B) Get conflicting graph
   uint32_t *conflict_vertices(NULL), *conflict_offset(NULL);   
   HANDLE_ERROR(hipMallocManaged(&conflict_offset, ((*numColor) +2)*sizeof(uint32_t)));
   HANDLE_ERROR(hipMallocManaged(&conflict_vertices, NumRow*sizeof(uint32_t)));   
   HANDLE_ERROR(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,temp_storage_bytes, numberVerticesPerColor,conflict_offset, (*numColor)+2));
   HANDLE_ERROR(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));   
   HANDLE_ERROR(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,temp_storage_bytes, numberVerticesPerColor,conflict_offset, (*numColor)+2));
   hipDeviceSynchronize();
   HANDLE_ERROR(hipFree(numberVerticesPerColor));
   uint32_t items_in_sh_mem = min(40*1024, NumRow);//same as bytes of sh mem since colors are unisgned char 
   conflict_graph<<< 1, (*numColor), items_in_sh_mem>>> (NumRow, color, numColor,conflict_vertices, conflict_offset, items_in_sh_mem);   
   hipDeviceSynchronize();     





   //C) Resolving conflict 
   uint32_t *lowTr_col(NULL), *lowTr_offset(NULL);
   HANDLE_ERROR(hipMallocManaged(&lowTr_col, numNNZ/2*sizeof(uint32_t)));
   HANDLE_ERROR(hipMallocManaged(&lowTr_offset, (NumRow+1)*sizeof(uint32_t)));
   getLowTrCSR(numNNZ, NumRow, graph, lowTr_col, lowTr_offset);

   int newNumColor = (*numColor);
   for(int i=1; i <= (*numColor); i++){
      newNumColor = conflict_resolve_forgetabout_sharedmemory1(conflict_vertices, conflict_offset, lowTr_col, lowTr_offset, NumRow, numNNZ/2, newNumColor, color, i, numBlocks, numThreads);
   }

   hipDeviceSynchronize();
   /***********************************************************************/

    //5.5) Validate parallel LOCAL solution    
   //printf("Parallel LOCAL solution has %d colors\n", CountColors(V, color));
   //printf("Valid LOCAL coloring: %d\n\n", IsValidColoring_Blocked(graph, V, color, blockingSize));  
   //PrintSolution(color,V);  
   //exit(0);   

   //6) Validate parallel solution 
   //printf("Parallel solution has %d colors\n", CountColors(V, color));
   //printf("Valid coloring: %d\n\n", IsValidColoring(graph, V, color));
   //PrintSolution(color,V);

   color_int = new int*[NumRow];
   for(uint32_t i=0;i<NumRow;i++){
      color_int[i]=new int[1];
      color_int[i][0] = int(color[i]);
   }
}


int main(int argc, char* argv[]){

   cuda_query(3); //Set the deivde number here 

   if(argc != 2){
      std::cout<<"  Usage ./graphGPU INPUTFILE"<<std::endl;
      std::cout<<"input files can be found under input/ "<<std::endl;
      exit(EXIT_FAILURE);
   }

   int** color = NULL;

   GraphColoringGPU(argv[1],color);

   /*for(int i=0;i<12;i++){
      std::cout<< " color["<< i<<"][0]= "<< color[i][0]<<std::endl;
   }*/


   /*bool* graph;
   int V;     
   const uint32_t blockingSize = 5;//TODO
   uint32_t numNNZ=0;
   uint32_t NumRow=0; 
   uint32_t numNNZ_blocked = 0;


   //1) Read graph
   if (std::string(argv[1]).find(".col") != std::string::npos){
     ReadColFile(argv[1], &graph, &V, &numNNZ,&NumRow, blockingSize, &numNNZ_blocked);
   } else if (std::string(argv[1]).find(".mm") != std::string::npos){
     ReadMMFile(argv[1], &graph, &V, &numNNZ,&NumRow,blockingSize, &numNNZ_blocked);
   } else{
      std::cout<<" Invalid file formate!!"<<std::endl;
      exit(EXIT_FAILURE);
   }      
  

   //2) Allocate memory (on both sides)
   uint32_t *col_id(NULL),*offset(NULL);   

   HANDLE_ERROR(hipMallocManaged(&col_id, numNNZ_blocked*sizeof(uint32_t)));   
   HANDLE_ERROR(hipMallocManaged(&offset, (NumRow +1)*sizeof(uint32_t)));   


   unsigned char* color;
   HANDLE_ERROR(hipMallocManaged(&color, NumRow*sizeof(unsigned char)));   
   memset(color, 0, NumRow);   
   int*numColor;
   HANDLE_ERROR(hipMallocManaged(&numColor, sizeof(int)));
   memset(numColor, 0, 1);   
   uint32_t*numberVerticesPerColor;//allocate as if each vertex will take different color 
   HANDLE_ERROR(hipMallocManaged(&numberVerticesPerColor, NumRow*sizeof(uint32_t)));
   memset(numberVerticesPerColor, 0, NumRow);   
  
   

   //3) Get graph in CSR format 
   //getCSR(numNNZ, NumRow, graph, col_id, offset);   
   uint32_t maxLeftout=0; //maxLeftout the maximum number of vertices j connected to i that are left out when constructing blocked CSR (used to allocate conflicting graph)
   getBlockedCSR(NumRow, graph, col_id, offset, blockingSize, maxLeftout);   
   //printCSR(numNNZ_blocked,NumRow,col_id, offset);   
   //exit(0);
   //getLowTrCSR(numNNZ, NumRow, graph, lowTr_col, lowTr_offset);
   //printCSR(numNNZ/2, NumRow, lowTr_col, lowTr_offset);
   

   //CUB parameters 
   void *d_temp_storage = NULL;
   size_t temp_storage_bytes = 0;
   hipcub::CachingDeviceAllocator  g_allocator(true);

   int numBlocks(1), numThreads(1);
   numBlocks = (NumRow/blockingSize) + 1;
   numThreads = blockingSize;
   //if(blockingSize < 1024){//if it is less than NumRow vertex, then launch one block 
   //   numBlocks = 1;
   //   numThreads = blockingSize;
   //}else{//otherwise, launch as many as 1024-blocks as you need      
   //   numBlocks = (blockingSize/1024) + 1;
   //   numThreads = 1024;
   //}


   //A) Do local colring 
   uint32_t max_NNZ_per_block= maxNNZ_per_segment(offset, NumRow, blockingSize);        
   uint32_t shrd_mem = numThreads*sizeof(bool);
   //+ max_NNZ_per_block*sizeof(uint32_t);  
   //std::cout<<"NumRow= "<<NumRow <<"   max_NNZ_per_block= "<<max_NNZ_per_block << " shrd_mem= "<<shrd_mem  << std::endl;
   //std::cout<<" numThreads= "<<numThreads<< " numBlocks= "<< numBlocks<<std::endl;
   coloring <<<numBlocks, numThreads, shrd_mem>>> (NumRow, col_id, offset, color, numColor,numberVerticesPerColor, max_NNZ_per_block);
   hipDeviceSynchronize();     

   HANDLE_ERROR(hipFree(offset));//free what you dont need 
   HANDLE_ERROR(hipFree(col_id));

   //B) Get conflicting graph
   uint32_t *conflict_vertices(NULL), *conflict_offset(NULL);   
   HANDLE_ERROR(hipMallocManaged(&conflict_offset, ((*numColor) +2)*sizeof(uint32_t)));
   HANDLE_ERROR(hipMallocManaged(&conflict_vertices, NumRow*sizeof(uint32_t)));   
   HANDLE_ERROR(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,temp_storage_bytes, numberVerticesPerColor,conflict_offset, (*numColor)+2));
   HANDLE_ERROR(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));   
   HANDLE_ERROR(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,temp_storage_bytes, numberVerticesPerColor,conflict_offset, (*numColor)+2));
   hipDeviceSynchronize();
   HANDLE_ERROR(hipFree(numberVerticesPerColor));
   uint32_t items_in_sh_mem = min(40*1024, NumRow);//same as bytes of sh mem since colors are unisgned char 
   conflict_graph<<< 1, (*numColor), items_in_sh_mem>>> (NumRow, color, numColor,conflict_vertices, conflict_offset, items_in_sh_mem);   
   hipDeviceSynchronize();     

   //C) Resolving conflict 
   uint32_t *lowTr_col(NULL), *lowTr_offset(NULL);
   HANDLE_ERROR(hipMallocManaged(&lowTr_col, numNNZ/2*sizeof(uint32_t)));
   HANDLE_ERROR(hipMallocManaged(&lowTr_offset, (NumRow+1)*sizeof(uint32_t)));
   getLowTrCSR(numNNZ, NumRow, graph, lowTr_col, lowTr_offset);

//   for(int index=0; index<NumRow; index++)
//   {
//	if(graph[100*NumRow+index]!=0)
//   	std::cout<<"graph[100*NumRow+"<<index<<"]:"<<graph[100*NumRow+index]<<" ";
//   }
//   for(int index=0; index<10; index++)
//   {
//   	int nodenow = lowTr_offset[index+1]-lowTr_offset[index];
//	std::cout<<"index: "<<index<<std::endl;
//   	for(int i=0; i<nodenow; i++)
//   	{
//		std::cout<<i<<":"<<lowTr_col[lowTr_offset[index]+i]<<"  ";
//   	}
//   	std::cout<<std::endl;
//   }


//   int node149Len = lowTr_offset[149+1]-lowTr_offset[149];
//   for(int i=0; i<node149Len; i++)
//   {
//	std::cout<<i<<":"<<lowTr_col[lowTr_offset[149]+i]<<"  ";
//   }
//   std::cout<<std::endl;

   int newNumColor = (*numColor);
   for(int i=1; i <= (*numColor); i++){
      newNumColor = conflict_resolve_forgetabout_sharedmemory1(conflict_vertices, conflict_offset, lowTr_col, lowTr_offset, NumRow, numNNZ/2, newNumColor, color, i, numBlocks, numThreads);
   }

   hipDeviceSynchronize();


    //5.5) Validate parallel LOCAL solution    
   //printf("Parallel LOCAL solution has %d colors\n", CountColors(V, color));
   //printf("Valid LOCAL coloring: %d\n\n", IsValidColoring_Blocked(graph, V, color, blockingSize));  
   //PrintSolution(color,V);  
   //exit(0);   

   printf("\n*********************************\n");
   //int* colorInt;
   //HANDLE_ERROR(hipMallocManaged(&colorInt, NumRow*sizeof(int)));   
   //memset(colorInt, 0, NumRow);   

   //6) Validate parallel solution 
   printf("Parallel solution has %d colors\n", CountColors(V, color));
   printf("Valid coloring: %d\n\n", IsValidColoring(graph, V, color));
   //PrintSolution(color,V);  

    //GreedyColoring(graph, V, &color);
   //printf("\n*********************************\n");
   //printf("Greedy solution has %d colors\n", CountColors(V, colorInt));
   //printf("Valid coloring: %d\n\n", IsValidColoring(graph, V, colorInt));
   //PrintSolution(colorInt,V);


   //printf("\n*********************************\n");
   //7) Color Vertices on CPU
   //GraphColoring(graph, V, &colorInt);
   //printf("Brute-foce solution has %d colors\n", CountColors(V, colorInt));   
   //printf("Valid coloring: %d\n", IsValidColoring(graph, V, colorInt));  
   */


   return 0;
}
