#include "hip/hip_runtime.h"
#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_load_balance.hxx>

using namespace mgpu;

using namespace hipcub;

__global__
void GetneighLen(uint32_t *nodes, int sizeNode,  uint32_t *tr_offset, uint32_t *neighLen, uint32_t *changeColor){
	
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeNode; i+=gridDim.x*blockDim.x)
	{
		neighLen[i] = tr_offset[nodes[i]+1]-tr_offset[nodes[i]];
		changeColor[i] = 0;
	}
}



__global__
void FindChangeColor(uint32_t *changeColor, uint32_t sizeNode, uint32_t *nodes, int *wir, int *lbs, uint32_t sizeLbs, uint32_t *tr_col_id, uint32_t *tr_offset, int theColor, unsigned char *color) {
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeLbs; i+=blockDim.x*gridDim.x)
	{
		int neighborOwner = lbs[i];
		int neighbor = tr_col_id[tr_offset[nodes[lbs[i]]] + wir[i]];
//		if(threadIdx.x==0)
//		{
//			for(int x=0; x<sizeNode; x++)
//				printf("nodes[%d]=%d  ", x, nodes[i]);
//			printf("\n");
//			printf("theColor: %u\n", (unsigned char)theColor);
//		}
//			printf("\n");
//		if(i==0)
//		printf("theColor: %u\n", (unsigned char)theColor);
//		if(i>=0 && i<=20) {
//			printf("thread %d, lbs[%d] is %d, wir[%d] is %d, nodes[lbs[%d]] is %d, my neighbor is %d, neighborOwner is %d, \n", i, i, lbs[i],i, wir[i], i, nodes[lbs[i]], neighbor, neighborOwner);
//			printf("thread %d, nodes[%d] is %d and color[%d] is %u\n", i, neighborOwner, nodes[neighborOwner], neighbor, color[neighbor]);
//		}
		
		if(color[neighbor] == (unsigned char)theColor)
		{
			changeColor[neighborOwner]=1;
		}
	}
//	if(threadIdx.x==0)
//                {
//                        for(int i=0; i<sizeNode; i++)
//                                printf("changeColor[%d]=%d   ", i,changeColor[i] );
//                        printf("\n");
//                }

}

__global__
void Conflict_assignColor(uint32_t *changeColor, int theColor, unsigned char *color, uint32_t *nodes, uint32_t sizeNode) {
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeNode; i+=blockDim.x*gridDim.x)
	{
		if(changeColor[i]==1)
			color[nodes[i]] = (unsigned char)theColor;
	}
}


__global__
void GenNewNodes(uint32_t *nodes, uint32_t *newNodes, uint32_t *neighLen, uint32_t *newNeighLen, int sizeNode, int *changeColor) {

	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeNode; i+=blockDim.x*gridDim.x)
	{
		if(changeColor[i]+1 == changeColor[i+1])
		{
			newNodes[changeColor[i]] = nodes[i];
			newNeighLen[changeColor[i]] = neighLen[i];
		}
	}
}

__global__
void WorkItemRank(int *scan, int *lbs, int *wir, int sizeLbs) {
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeLbs; i+=blockDim.x*gridDim.x)
	{
		wir[i] = i - scan[lbs[i]];
	}
}

__global__
void ResetChangeColor(int sizeNode, uint32_t *changeColor) {
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<sizeNode; i+=blockDim.x*gridDim.x)
	{
		changeColor[i]=0;
	}
}



int conflict_resolve_forgetabout_sharedmemory1(uint32_t* conflict_color, // Array of conflict vetices grouped by color
                      uint32_t *conflict_color_offset, // offset of different color on conflit_color 
                      uint32_t *tr_col_id, // CSR of graph, but only lower triangle part
                      uint32_t *tr_offset, // CSR offset of graph, but only lower triangle part
                      uint32_t numVertices, // number of vertices
                      uint32_t size_tr_col,// size of tr_col_id
                      uint32_t numColor,// number of color has been used
                      unsigned char *color,// color array for all vertices
		      uint32_t colorID,
		      int gridSize,
		      int blockSize ) // working space and the size of this array shoudl be BLOCK_THREADS*ITEM_PER_THREAD, o,w it overwrittern information
{
        standard_context_t context;
	uint32_t *nodes(NULL), *changeColor(NULL), *nodes1(NULL), *nodes2(NULL), *neighLen1(NULL), *neighLen2(NULL),  *neighLen(NULL);
	uint32_t *newNodes(NULL), *newNeighLen(NULL);
	int start = conflict_color_offset[colorID];
	int end = conflict_color_offset[colorID+1];
	int sizeNode = end-start;
	if(sizeNode==0) return numColor;
//	std::cout<<"start: "<<start<<" end: "<<end<<" sizeNode: "<<sizeNode <<std::endl;
	nodes = conflict_color+start;
	HANDLE_ERROR(hipMallocManaged(&changeColor, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&nodes1, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&nodes2, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&neighLen1, sizeNode*sizeof(uint32_t)));
	HANDLE_ERROR(hipMallocManaged(&neighLen2, sizeNode*sizeof(uint32_t)));
        int *lbs(NULL), *wir(NULL);
	int *scanArray(NULL);
	HANDLE_ERROR(hipMallocManaged(&scanArray, (sizeNode+1)*sizeof(int)));
	
//	std::cout<<"allocate succeed"<<std::endl;
//	for(int i=0; i<sizeNode; i++)
//	{
//		std::cout<<"nodes["<<i<<"]= "<<nodes[i]<<"  ";
//	}
//	std::cout<<std::endl;
//	std::cout<<std::endl;

	GetneighLen<<<gridSize, blockSize>>>(nodes, sizeNode,  tr_offset, neighLen1, changeColor);
	hipDeviceSynchronize();



//	for(int i=0; i<sizeNode; i++)
//	{
//		std::cout<<"changeColor["<<i<<"]= "<<changeColor[i]<<"  ";
//	}
//	std::cout<<std::endl;
//		for(int i=0; i<numVertices; i++)
//		{
//			printf("color[%d]=%u  ", i, color[i]);
//		}
//		std::cout<<std::endl;

	int choseL = 0;
	int counter = 1;
	neighLen = neighLen1;
	int theColor = colorID;
	void  *d_temp_storage = NULL;
    	size_t    temp_storage_bytes = 0;
	DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, neighLen, scanArray, sizeNode+1);
	HANDLE_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));

	while(true) {
//		for(int i=0; i<sizeNode; i++)
//		{
//			std::cout<<"neighLen["<<i<<"]= "<<neighLen[i]<<"  ";
//		}
//		std::cout<<std::endl;
//		std::cout<<std::endl;

		DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, neighLen, scanArray, sizeNode+1);			
		hipDeviceSynchronize();
//		for(int i=0; i<sizeNode+1; i++)
//		{
//			std::cout<<"scan_neighLen["<<i<<"]= "<<scanArray[i]<<"   ";
//		}
//		std::cout<<std::endl;
//		std::cout<<std::endl;
		
		int sizeLbs = scanArray[sizeNode];

		if(counter == 1) {
        		HANDLE_ERROR(hipMallocManaged(&lbs, sizeLbs*sizeof(int)));
        		HANDLE_ERROR(hipMallocManaged(&wir, sizeLbs*sizeof(int)));
		}

		load_balance_search(sizeLbs, scanArray, sizeNode, lbs, context);
		hipDeviceSynchronize();
//		for(int i=0; i<sizeLbs; i++)
//		{
//			std::cout<<"lbs["<<i<<"]= "<<lbs[i]<<"   ";
//		}
//		std::cout<<std::endl;
//		std::cout<<std::endl;
		
		WorkItemRank<<<gridSize,blockSize>>>(scanArray, lbs, wir, sizeLbs);		
		hipDeviceSynchronize();
//		for(int i=0; i<sizeLbs; i++)
//		{
//			std::cout<<"WIR["<<i<<"]= "<<wir[i]<<"   ";
//                }
//		std::cout<<std::endl;
//		std::cout<<std::endl;
		
                FindChangeColor<<<gridSize,blockSize>>>(changeColor, sizeNode, nodes, wir, lbs, sizeLbs, tr_col_id, tr_offset, theColor, color);
		hipDeviceSynchronize();

		theColor = numColor+counter;
		Conflict_assignColor<<<gridSize,blockSize>>>(changeColor, theColor, color, nodes, sizeNode);
		hipDeviceSynchronize();

//		for(int i=0; i<numVertices; i++)
//		{
//			 printf("color[%d]=%u  ", i, color[i]);
//		}
//		std::cout<<std::endl;
//		std::cout<<std::endl;

		DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, changeColor, scanArray, sizeNode+1);	
		hipDeviceSynchronize();

//		for(int i=0; i<sizeNode; i++)
//		{
//			std::cout<<"changeColor["<<i<<"]= "<<changeColor[i]<<"  ";
//		}
//		std::cout<<std::endl;	
//		std::cout<<std::endl;
		
//		for(int i=0; i<sizeNode+1; i++)
//		{
//			std::cout<<"scan_changeColor["<<i<<"]= "<<scanArray[i]<<"   ";
//		}
//		std::cout<<std::endl;
//		std::cout<<std::endl;

		choseL = choseL^1;
		if(choseL == 1)
		{
			newNeighLen = neighLen2;
			newNodes = nodes2;
		}
		else
		{
			newNeighLen = neighLen1;
			newNodes = nodes1;
		}
	
//		std::cout<<"sizeNode: "<< sizeNode<<std::endl;
		GenNewNodes<<<gridSize,blockSize>>>(nodes, newNodes, neighLen, newNeighLen, sizeNode, scanArray);
		hipDeviceSynchronize();
		sizeNode = scanArray[sizeNode];
		ResetChangeColor<<<gridSize,blockSize>>>(sizeNode, changeColor);
		hipDeviceSynchronize();
		if(sizeNode == 0) break;

		printf("new sizeNode: %d\n", sizeNode);
		nodes = newNodes;
		for(int i=0; i<sizeNode; i++)
		{
			std::cout<<"newNode["<<i<<"]=  "<<nodes[i]<<"   ";
		}
		std::cout<<std::endl;
		std::cout<<std::endl;
		neighLen = newNeighLen;
//		for(int i=0; i<sizeNode; i++)
//		{
//			std::cout<<"newNeighLen["<<i<<"]=  "<<neighLen[i]<<"   ";
//		}
//		std::cout<<std::endl;
//		std::cout<<std::endl;
//		std::cout<<"counter: "<<counter<<std::endl;
		counter++;
//		std::cout<<std::endl;
//		std::cout<<std::endl;
	}	
	std::cout<<counter-1<<" color is added, total number of color is "<<theColor-1<<std::endl;
	hipFree(nodes1);
	hipFree(nodes2);
	hipFree(neighLen1);
	hipFree(neighLen2);
	hipFree(changeColor);
	hipFree(scanArray);
	hipFree(d_temp_storage);
	hipFree(lbs);
	hipFree(wir);
	return theColor-1;
}
